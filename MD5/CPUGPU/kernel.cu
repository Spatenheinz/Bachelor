#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include "cudaHelpers.cu.h"
#include "kernel.cu.h"

// Bad globals - not ideal
uint32_t *dev_k = 0;
uint32_t *dev_r = 0;

#define BLOCKS 512
__device__ void d_to_bytes(uint32_t val, uint8_t *bytes)
{
    bytes[0] = (uint8_t) val;
    bytes[1] = (uint8_t) (val >> 8);
    bytes[2] = (uint8_t) (val >> 16);
    bytes[3] = (uint8_t) (val >> 24);
}

__device__ uint32_t cuda_to_int32(const uint8_t *bytes)
{
    return (uint32_t) bytes[0]
        | ((uint32_t) bytes[1] << 8)
        | ((uint32_t) bytes[2] << 16)
        | ((uint32_t) bytes[3] << 24);
}

__global__ void md5kernel(const uint8_t *initial_msg, size_t initial_len, uint8_t *digest) {
    // These vars will contain the hash
    uint32_t A, B, C, D;
    // Message (to prepare)
    uint8_t *msg = NULL;
    size_t new_len, offset;
    uint32_t w[16];
    uint32_t a, b, c, d, i;
    // Initialize variables - simple count in nibbles:
    A = 0x67452301;
    B = 0xefcdab89;
    C = 0x98badcfe;
    D = 0x10325476;
    //Pre-processing:
    //append "1" bit to message
    //append "0" bits until message length in bits ≡ 448 (mod 512)
    //append length mod (2^64) to message
    for (new_len = initial_len + 1; new_len % (512/8) != 448/8; new_len++)
        ;
    msg = (uint8_t *)malloc(new_len + 8);
    #pragma unroll
    for(int i = 0; i < initial_len; i++) {
        msg[i] = initial_msg[i];
    }
    // memcpy(msg, initial_msg, initial_len);
    msg[initial_len] = 0x80; // append the "1" bit; most significant bit is "first"
    #pragma unroll
    for (offset = initial_len + 1; offset < new_len; offset++)
        msg[offset] = 0; // append "0" bits
    // append the len in bits at the end of the buffer.
    d_to_bytes(initial_len*8, msg + new_len);
    // initial_len>>29 == initial_len*8>>32, but avoids overflow.
    d_to_bytes(initial_len>>29, msg + new_len + 4);
    // Process the message in successive 512-bit chunks:
    //for each 512-bit chunk of message:
    #pragma unroll
    for(offset=0; offset<new_len; offset += (512/8)) {
        // break chunk into sixteen 32-bit words w[j], 0 ≤ j ≤ 15
        #pragma unroll
        for (i = 0; i < 16; i++)
            w[i] = cuda_to_int32(msg + offset + i*4);
        // Initialize hash value for this chunk:
        a = A;
        b = B;
        c = C;
        d = D;
        // Main loop:
        FF ( a, b, c, d, w[ 0], 7, 3614090360);  FF ( d, a, b, c, w[ 1], 12, 3905402710);
        FF ( c, d, a, b, w[ 2], 17,  606105819); FF ( b, c, d, a, w[ 3], 22, 3250441966);
        FF ( a, b, c, d, w[ 4], 7, 4118548399);  FF ( d, a, b, c, w[ 5], 12, 1200080426);
        FF ( c, d, a, b, w[ 6], 17, 2821735955); FF ( b, c, d, a, w[ 7], 22, 4249261313);
        FF ( a, b, c, d, w[ 8], 7, 1770035416);  FF ( d, a, b, c, w[ 9], 12, 2336552879);
        FF ( c, d, a, b, w[10], 17, 4294925233); FF ( b, c, d, a, w[11], 22, 2304563134);
        FF ( a, b, c, d, w[12], 7, 1804603682);  FF ( d, a, b, c, w[13], 12, 4254626195);
        FF ( c, d, a, b, w[14], 17, 2792965006); FF ( b, c, d, a, w[15], 22, 1236535329);
        GG ( a, b, c, d, w[ 1], 5, 4129170786);  GG ( d, a, b, c, w[ 6], 9, 3225465664);
        GG ( c, d, a, b, w[11], 14,  643717713); GG ( b, c, d, a, w[ 0], 20, 3921069994);
        GG ( a, b, c, d, w[ 5], 5, 3593408605);  GG ( d, a, b, c, w[10], 9,   38016083);
        GG ( c, d, a, b, w[15], 14, 3634488961); GG ( b, c, d, a, w[ 4], 20, 3889429448);
        GG ( a, b, c, d, w[ 9], 5,  568446438);  GG ( d, a, b, c, w[14], 9, 3275163606);
        GG ( c, d, a, b, w[ 3], 14, 4107603335); GG ( b, c, d, a, w[ 8], 20, 1163531501);
        GG ( a, b, c, d, w[13], 5, 2850285829);  GG ( d, a, b, c, w[ 2], 9, 4243563512);
        GG ( c, d, a, b, w[ 7], 14, 1735328473); GG ( b, c, d, a, w[12], 20, 2368359562);
        HH ( a, b, c, d, w[ 5], 4, 4294588738);  HH ( d, a, b, c, w[ 8], 11, 2272392833);
        HH ( c, d, a, b, w[11], 16, 1839030562); HH ( b, c, d, a, w[14], 23, 4259657740);
        HH ( a, b, c, d, w[ 1], 4, 2763975236);  HH ( d, a, b, c, w[ 4], 11, 1272893353);
        HH ( c, d, a, b, w[ 7], 16, 4139469664); HH ( b, c, d, a, w[10], 23, 3200236656);
        HH ( a, b, c, d, w[13], 4,  681279174);  HH ( d, a, b, c, w[ 0], 11, 3936430074);
        HH ( c, d, a, b, w[ 3], 16, 3572445317); HH ( b, c, d, a, w[ 6], 23,   76029189);
        HH ( a, b, c, d, w[ 9], 4, 3654602809);  HH ( d, a, b, c, w[12], 11, 3873151461);
        HH ( c, d, a, b, w[15], 16,  530742520); HH ( b, c, d, a, w[ 2], 23, 3299628645);
        II ( a, b, c, d, w[ 0], 6, 4096336452);  II ( d, a, b, c, w[ 7], 10, 1126891415);
        II ( c, d, a, b, w[14], 15, 2878612391); II ( b, c, d, a, w[ 5], 21, 4237533241);
        II ( a, b, c, d, w[12], 6, 1700485571);  II ( d, a, b, c, w[ 3], 10, 2399980690);
        II ( c, d, a, b, w[10], 15, 4293915773); II ( b, c, d, a, w[ 1], 21, 2240044497);
        II ( a, b, c, d, w[ 8], 6, 1873313359);  II ( d, a, b, c, w[15], 10, 4264355552);
        II ( c, d, a, b, w[ 6], 15, 2734768916); II ( b, c, d, a, w[13], 21, 1309151649);
        II ( a, b, c, d, w[ 4], 6, 4149444226);  II ( d, a, b, c, w[11], 10, 3174756917);
        II ( c, d, a, b, w[ 2], 15,  718787259); II ( b, c, d, a, w[ 9], 21, 3951481745);
        // Add this chunk's hash to result so far:
        A += a;
        B += b;
        C += c;
        D += d;
    }

    // cleanup
    free(msg);
    //var char digest[16] := A append B append C append D //(Output is in little-endian)
    d_to_bytes(A, digest);
    d_to_bytes(B, digest + 4);
    d_to_bytes(C, digest + 8);
    d_to_bytes(D, digest + 12);
}
__global__ void many_md5kernel(const uint8_t *initial_msg, size_t initial_len, uint8_t *digest) {
    // These vars will contain the hash
    uint32_t A, B, C, D;
    // Message (to prepare)
    uint8_t *msg = NULL;
    size_t new_len, offset;
    uint32_t w[16];
    uint32_t a, b, c, d, i;
    // Initialize variables - simple count in nibbles:
    A = 0x67452301;
    B = 0xefcdab89;
    C = 0x98badcfe;
    D = 0x10325476;
    //Pre-processing:
    //append "1" bit to message
    //append "0" bits until message length in bits ≡ 448 (mod 512)
    //append length mod (2^64) to message
    initial_len /= gridDim.x;
    // printf("%d\n", initial_len);
    for (new_len = initial_len + 1; new_len % (512/8) != 448/8; new_len++)
        ;

    msg = (uint8_t *)malloc(new_len + 8);

    for(int i=0; i < initial_len; i++) {
        msg[i] = initial_msg[blockIdx.x*initial_len+i];
    }
    msg[initial_len] = 0x80; // append the "1" bit; most significant bit is "first"
    #pragma unroll
    for (offset = initial_len + 1; offset < new_len; offset++)
        msg[offset] = 0; // append "0" bits
    // append the len in bits at the end of the buffer.
    d_to_bytes(initial_len*8, msg + new_len);
    // initial_len>>29 == initial_len*8>>32, but avoids overflow.
    d_to_bytes(initial_len>>29, msg + new_len + 4);
    // Process the message in successive 512-bit chunks:
    //for each 512-bit chunk of message:
    #pragma unroll
    for(offset=0; offset<new_len; offset += (512/8)) {
        // break chunk into sixteen 32-bit words w[j], 0 ≤ j ≤ 15
        #pragma unroll
        for (i = 0; i < 16; i++)
            w[i] = cuda_to_int32(msg + offset + i*4);
        // Initialize hash value for this chunk:
        a = A;
        b = B;
        c = C;
        d = D;
        // Main loop:
        FF ( a, b, c, d, w[ 0], 7, 3614090360);  FF ( d, a, b, c, w[ 1], 12, 3905402710);
        FF ( c, d, a, b, w[ 2], 17,  606105819); FF ( b, c, d, a, w[ 3], 22, 3250441966);
        FF ( a, b, c, d, w[ 4], 7, 4118548399);  FF ( d, a, b, c, w[ 5], 12, 1200080426);
        FF ( c, d, a, b, w[ 6], 17, 2821735955); FF ( b, c, d, a, w[ 7], 22, 4249261313);
        FF ( a, b, c, d, w[ 8], 7, 1770035416);  FF ( d, a, b, c, w[ 9], 12, 2336552879);
        FF ( c, d, a, b, w[10], 17, 4294925233); FF ( b, c, d, a, w[11], 22, 2304563134);
        FF ( a, b, c, d, w[12], 7, 1804603682);  FF ( d, a, b, c, w[13], 12, 4254626195);
        FF ( c, d, a, b, w[14], 17, 2792965006); FF ( b, c, d, a, w[15], 22, 1236535329);
        GG ( a, b, c, d, w[ 1], 5, 4129170786);  GG ( d, a, b, c, w[ 6], 9, 3225465664);
        GG ( c, d, a, b, w[11], 14,  643717713); GG ( b, c, d, a, w[ 0], 20, 3921069994);
        GG ( a, b, c, d, w[ 5], 5, 3593408605);  GG ( d, a, b, c, w[10], 9,   38016083);
        GG ( c, d, a, b, w[15], 14, 3634488961); GG ( b, c, d, a, w[ 4], 20, 3889429448);
        GG ( a, b, c, d, w[ 9], 5,  568446438);  GG ( d, a, b, c, w[14], 9, 3275163606);
        GG ( c, d, a, b, w[ 3], 14, 4107603335); GG ( b, c, d, a, w[ 8], 20, 1163531501);
        GG ( a, b, c, d, w[13], 5, 2850285829);  GG ( d, a, b, c, w[ 2], 9, 4243563512);
        GG ( c, d, a, b, w[ 7], 14, 1735328473); GG ( b, c, d, a, w[12], 20, 2368359562);
        HH ( a, b, c, d, w[ 5], 4, 4294588738);  HH ( d, a, b, c, w[ 8], 11, 2272392833);
        HH ( c, d, a, b, w[11], 16, 1839030562); HH ( b, c, d, a, w[14], 23, 4259657740);
        HH ( a, b, c, d, w[ 1], 4, 2763975236);  HH ( d, a, b, c, w[ 4], 11, 1272893353);
        HH ( c, d, a, b, w[ 7], 16, 4139469664); HH ( b, c, d, a, w[10], 23, 3200236656);
        HH ( a, b, c, d, w[13], 4,  681279174);  HH ( d, a, b, c, w[ 0], 11, 3936430074);
        HH ( c, d, a, b, w[ 3], 16, 3572445317); HH ( b, c, d, a, w[ 6], 23,   76029189);
        HH ( a, b, c, d, w[ 9], 4, 3654602809);  HH ( d, a, b, c, w[12], 11, 3873151461);
        HH ( c, d, a, b, w[15], 16,  530742520); HH ( b, c, d, a, w[ 2], 23, 3299628645);
        II ( a, b, c, d, w[ 0], 6, 4096336452);  II ( d, a, b, c, w[ 7], 10, 1126891415);
        II ( c, d, a, b, w[14], 15, 2878612391); II ( b, c, d, a, w[ 5], 21, 4237533241);
        II ( a, b, c, d, w[12], 6, 1700485571);  II ( d, a, b, c, w[ 3], 10, 2399980690);
        II ( c, d, a, b, w[10], 15, 4293915773); II ( b, c, d, a, w[ 1], 21, 2240044497);
        II ( a, b, c, d, w[ 8], 6, 1873313359);  II ( d, a, b, c, w[15], 10, 4264355552);
        II ( c, d, a, b, w[ 6], 15, 2734768916); II ( b, c, d, a, w[13], 21, 1309151649);
        II ( a, b, c, d, w[ 4], 6, 4149444226);  II ( d, a, b, c, w[11], 10, 3174756917);
        II ( c, d, a, b, w[ 2], 15,  718787259); II ( b, c, d, a, w[ 9], 21, 3951481745);
        // Add this chunk's hash to result so far:
        A += a;
        B += b;
        C += c;
        D += d;
    }

    // cleanup
    free(msg);
    //var char digest[16] := A append B append C append D //(Output is in little-endian)
    d_to_bytes(A, digest + (blockIdx.x*md5_size));
    d_to_bytes(B, digest + (blockIdx.x*md5_size) + 4);
    d_to_bytes(C, digest + (blockIdx.x*md5_size) + 8);
    d_to_bytes(D, digest + (blockIdx.x*md5_size) + 12);
}

// Helper function for using CUDA to compute MD5 with timing
int MD5(const uint8_t *initial_msg, size_t initial_len, uint8_t *digest, int runs)
{
    struct timeval t_start, t_end, t_diff;
    uint8_t *dev_initial_msg = 0;
    uint8_t *dev_digest = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    gpuErrchk(hipSetDevice(0));
    gpuErrchk(hipMalloc((void**)&dev_digest, md5_size * sizeof(uint8_t)));
    gpuErrchk(hipMalloc((void**)&dev_initial_msg, initial_len * sizeof(uint8_t)));
    // Copy input vectors from host memory to GPU buffers.
    gpuErrchk(hipMemcpy(dev_initial_msg, initial_msg, initial_len * sizeof(uint8_t), hipMemcpyHostToDevice));
	gettimeofday(&t_start, NULL);
    for (int i = 0; i < runs; i++) {
        md5kernel<<<1, 1>>>(dev_initial_msg, initial_len, dev_digest);
    }
    gpuErrchk(hipDeviceSynchronize());
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        float elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / runs;
        double MBsec = initial_len / elapsed;
        printf("CPU runs in:       %.0f microsecs,   MB/sec: %.2f\n", elapsed, MBsec);

    // Check for any errors launching the kernel
    gpuErrchk(hipGetLastError());
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.

    // Copy output vector from GPU buffer to host memory.
    gpuErrchk(hipMemcpy(digest, dev_digest, md5_size * sizeof(uint8_t), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(dev_digest));
    gpuErrchk(hipFree(dev_initial_msg));

    return 0;
}



int Many_MD5(const uint8_t *initial_msg, size_t initial_len, uint8_t *digest, int runs)
{
    struct timeval t_start, t_end, t_diff;
    uint8_t *dev_initial_msg = 0;
    uint8_t *dev_digest = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    gpuErrchk(hipSetDevice(0));

    gpuErrchk(hipMalloc((void**)&dev_digest, md5_size * BLOCKS * sizeof(uint8_t)));
    gpuErrchk(hipMalloc((void**)&dev_initial_msg, initial_len * sizeof(uint8_t)));
    // Copy input vectors from host memory to GPU buffers.
    gpuErrchk(hipMemcpy(dev_initial_msg, initial_msg, initial_len * sizeof(uint8_t), hipMemcpyHostToDevice));

    // Launch a kernel on the GPU with one thread for each element.
	gettimeofday(&t_start, NULL);
    for (int i = 0; i < runs; i++) {
        many_md5kernel<<<BLOCKS, 1>>>(dev_initial_msg, initial_len, dev_digest);
    }
    gpuErrchk(hipDeviceSynchronize());
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        float elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / runs;
        double MBsec = initial_len / elapsed;
        printf("CPU runs in:       %.0f microsecs,   MB/sec: %.2f\n", elapsed, MBsec);

    // Check for any errors launching the kernel
    gpuErrchk(hipGetLastError());
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.

    gpuErrchk(hipMemcpy(digest, dev_digest + ((BLOCKS-1)*md5_size), md5_size * sizeof(uint8_t), hipMemcpyDeviceToHost));
    gpuErrchk(hipFree(dev_digest));
    gpuErrchk(hipFree(dev_initial_msg));

    return 0;
}
