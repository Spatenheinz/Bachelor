#include "hip/hip_runtime.h"
#include "kernel.cu.h"
#include ""

#include <stdio.h>
#include <sys/time.h>
#define DEBUG_INFO  true
uint32_t MAX_HWDTH;
uint32_t MAX_BLOCK;
uint32_t MAX_SHMEM;

hipDeviceProp_t prop;
void initHwd() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    hipGetDeviceProperties(&prop, 0);
    MAX_HWDTH = prop.maxThreadsPerMultiProcessor * prop.multiProcessorCount;
    MAX_BLOCK = prop.maxThreadsPerBlock;
    MAX_SHMEM = prop.sharedMemPerBlock;

    if (DEBUG_INFO) {
        printf("Device name: %s\n", prop.name);
        printf("Number of hardware threads: %d\n", MAX_HWDTH);
        printf("Max block size: %d\n", MAX_BLOCK);
        printf("Shared memory size: %d\n", MAX_SHMEM);
        puts("====");
    }
}

// Bad globals
uint32_t *dev_k = 0;
uint32_t *dev_r = 0;
int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1)
{
    unsigned int resolution=1000000;
    long int diff = (t2->tv_usec + resolution * t2->tv_sec) - (t1->tv_usec + resolution * t1->tv_sec);
    result->tv_sec = diff / resolution;
    result->tv_usec = diff % resolution;
    return (diff<0);
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
__device__ void cuda_to_bytes(uint32_t val, uint8_t *bytes)
{
    bytes[0] = (uint8_t) val;
    bytes[1] = (uint8_t) (val >> 8);
    bytes[2] = (uint8_t) (val >> 16);
    bytes[3] = (uint8_t) (val >> 24);
}

__device__ uint32_t cuda_to_int32(const uint8_t *bytes)
{
    return (uint32_t) bytes[0]
        | ((uint32_t) bytes[1] << 8)
        | ((uint32_t) bytes[2] << 16)
        | ((uint32_t) bytes[3] << 24);
}

__global__ void md5kernel(const uint8_t *initial_msg, size_t initial_len, uint8_t *digest) {
    // These vars will contain the hash
    uint32_t A, B, C, D;
    // Message (to prepare)
    uint8_t *msg = NULL;
    size_t new_len, offset;
    uint32_t w[16];
    uint32_t a, b, c, d, i;
    // Initialize variables - simple count in nibbles:
    A = 0x67452301;
    B = 0xefcdab89;
    C = 0x98badcfe;
    D = 0x10325476;
    //Pre-processing:
    //append "1" bit to message
    //append "0" bits until message length in bits ≡ 448 (mod 512)
    //append length mod (2^64) to message
    for (new_len = initial_len + 1; new_len % (512/8) != 448/8; new_len++)
        ;
    msg = (uint8_t *)malloc(new_len + 8);
    memcpy(msg, initial_msg, initial_len);
    msg[initial_len] = 0x80; // append the "1" bit; most significant bit is "first"
    #pragma unroll
    for (offset = initial_len + 1; offset < new_len; offset++)
        msg[offset] = 0; // append "0" bits
    // append the len in bits at the end of the buffer.
    cuda_to_bytes(initial_len*8, msg + new_len);
    // initial_len>>29 == initial_len*8>>32, but avoids overflow.
    cuda_to_bytes(initial_len>>29, msg + new_len + 4);
    // Process the message in successive 512-bit chunks:
    //for each 512-bit chunk of message:
    #pragma unroll
    for(offset=0; offset<new_len; offset += (512/8)) {
        // break chunk into sixteen 32-bit words w[j], 0 ≤ j ≤ 15
        #pragma unroll
        for (i = 0; i < 16; i++)
            w[i] = cuda_to_int32(msg + offset + i*4);
        // Initialize hash value for this chunk:
        a = A;
        b = B;
        c = C;
        d = D;
        // Main loop:
        FF ( a, b, c, d, w[ 0], 7, 3614090360); /* 1 */ FF ( d, a, b, c, w[ 1], 12, 3905402710); /* 2 */
        FF ( c, d, a, b, w[ 2], 17,  606105819); /* 3 */FF ( b, c, d, a, w[ 3], 22, 3250441966); /* 4 */
        FF ( a, b, c, d, w[ 4], 7, 4118548399); /* 5 */ FF ( d, a, b, c, w[ 5], 12, 1200080426); /* 6 */
        FF ( c, d, a, b, w[ 6], 17, 2821735955); /* 7 */FF ( b, c, d, a, w[ 7], 22, 4249261313); /* 8 */
        FF ( a, b, c, d, w[ 8], 7, 1770035416); /* 9 */ FF ( d, a, b, c, w[ 9], 12, 2336552879); /* 10 */
        FF ( c, d, a, b, w[10], 17, 4294925233); /* 11 */FF ( b, c, d, a, w[11], 22, 2304563134); /* 12 */
        FF ( a, b, c, d, w[12], 7, 1804603682); /* 13 */FF ( d, a, b, c, w[13], 12, 4254626195); /* 14 */
        FF ( c, d, a, b, w[14], 17, 2792965006); /* 15 */FF ( b, c, d, a, w[15], 22, 1236535329); /* 16 */
        GG ( a, b, c, d, w[ 1], 5, 4129170786); /* 17 */GG ( d, a, b, c, w[ 6], 9, 3225465664); /* 18 */
        GG ( c, d, a, b, w[11], 14,  643717713); /* 19 */GG ( b, c, d, a, w[ 0], 20, 3921069994); /* 20 */
        GG ( a, b, c, d, w[ 5], 5, 3593408605); /* 21 */GG ( d, a, b, c, w[10], 9,   38016083); /* 22 */
        GG ( c, d, a, b, w[15], 14, 3634488961); /* 23 */GG ( b, c, d, a, w[ 4], 20, 3889429448); /* 24 */
        GG ( a, b, c, d, w[ 9], 5,  568446438); /* 25 */ GG ( d, a, b, c, w[14], 9, 3275163606); /* 26 */
        GG ( c, d, a, b, w[ 3], 14, 4107603335); /* 27 */GG ( b, c, d, a, w[ 8], 20, 1163531501); /* 28 */
        GG ( a, b, c, d, w[13], 5, 2850285829); /* 29 */GG ( d, a, b, c, w[ 2], 9, 4243563512); /* 30 */
        GG ( c, d, a, b, w[ 7], 14, 1735328473); /* 31 */GG ( b, c, d, a, w[12], 20, 2368359562); /* 32 */
        HH ( a, b, c, d, w[ 5], 4, 4294588738); /* 33 */HH ( d, a, b, c, w[ 8], 11, 2272392833); /* 34 */
        HH ( c, d, a, b, w[11], 16, 1839030562); /* 35 */HH ( b, c, d, a, w[14], 23, 4259657740); /* 36 */
        HH ( a, b, c, d, w[ 1], 4, 2763975236); /* 37 */HH ( d, a, b, c, w[ 4], 11, 1272893353); /* 38 */
        HH ( c, d, a, b, w[ 7], 16, 4139469664); /* 39 */HH ( b, c, d, a, w[10], 23, 3200236656); /* 40 */
        HH ( a, b, c, d, w[13], 4,  681279174); /* 41 */HH ( d, a, b, c, w[ 0], 11, 3936430074); /* 42 */
        HH ( c, d, a, b, w[ 3], 16, 3572445317); /* 43 */HH ( b, c, d, a, w[ 6], 23,   76029189);/* 44 */
        HH ( a, b, c, d, w[ 9], 4, 3654602809); /* 45 */HH ( d, a, b, c, w[12], 11, 3873151461); /* 46 */
        HH ( c, d, a, b, w[15], 16,  530742520); /* 47 */HH ( b, c, d, a, w[ 2], 23, 3299628645); /* 48 */
        II ( a, b, c, d, w[ 0], 6, 4096336452); /* 49 */II ( d, a, b, c, w[ 7], 10, 1126891415); /* 50 */
        II ( c, d, a, b, w[14], 15, 2878612391); /* 51 */II ( b, c, d, a, w[ 5], 21, 4237533241); /* 52 */
        II ( a, b, c, d, w[12], 6, 1700485571); /* 53 */II ( d, a, b, c, w[ 3], 10, 2399980690); /* 54 */
        II ( c, d, a, b, w[10], 15, 4293915773); /* 55 */II ( b, c, d, a, w[ 1], 21, 2240044497); /* 56 */
        II ( a, b, c, d, w[ 8], 6, 1873313359); /* 57 */II ( d, a, b, c, w[15], 10, 4264355552); /* 58 */
        II ( c, d, a, b, w[ 6], 15, 2734768916); /* 59 */II ( b, c, d, a, w[13], 21, 1309151649); /* 60 */
        II ( a, b, c, d, w[ 4], 6, 4149444226); /* 61 */II ( d, a, b, c, w[11], 10, 3174756917); /* 62 */
        II ( c, d, a, b, w[ 2], 15,  718787259); /* 63 */II ( b, c, d, a, w[ 9], 21, 3951481745); /* 64 */
        // Add this chunk's hash to result so far:
        A += a;
        B += b;
        C += c;
        D += d;
    }

    // cleanup
    free(msg);
    //var char digest[16] := A append B append C append D //(Output is in little-endian)
    cuda_to_bytes(A, digest);
    cuda_to_bytes(B, digest + 4);
    cuda_to_bytes(C, digest + 8);
    cuda_to_bytes(D, digest + 12);
}
__global__ void many_md5kernel(const uint8_t *initial_msg, size_t initial_len, uint8_t *digest) {
    // These vars will contain the hash
    uint32_t A, B, C, D;
    // Message (to prepare)
    uint8_t *msg = NULL;
    size_t new_len, offset;
    uint32_t w[16];
    uint32_t a, b, c, d, i;
    // Initialize variables - simple count in nibbles:
    A = 0x67452301;
    B = 0xefcdab89;
    C = 0x98badcfe;
    D = 0x10325476;
    //Pre-processing:
    //append "1" bit to message
    //append "0" bits until message length in bits ≡ 448 (mod 512)
    //append length mod (2^64) to message
    initial_len /= gridDim.x;
    for (new_len = initial_len + 1; new_len % (512/8) != 448/8; new_len++)
        ;

    msg = (uint8_t *)malloc(new_len + 8);
    memcpy(msg, (initial_msg+blockIdx.x*gridDim.x), initial_len);
    msg[initial_len] = 0x80; // append the "1" bit; most significant bit is "first"
    #pragma unroll
    for (offset = initial_len + 1; offset < new_len; offset++)
        msg[offset] = 0; // append "0" bits
    // append the len in bits at the end of the buffer.
    cuda_to_bytes(initial_len*8, msg + new_len);
    // initial_len>>29 == initial_len*8>>32, but avoids overflow.
    cuda_to_bytes(initial_len>>29, msg + new_len + 4);
    // Process the message in successive 512-bit chunks:
    //for each 512-bit chunk of message:
    #pragma unroll
    for(offset=0; offset<new_len; offset += (512/8)) {
        // break chunk into sixteen 32-bit words w[j], 0 ≤ j ≤ 15
        #pragma unroll
        for (i = 0; i < 16; i++)
            w[i] = cuda_to_int32(msg + offset + i*4);
        // Initialize hash value for this chunk:
        a = A;
        b = B;
        c = C;
        d = D;
        // Main loop:
        FF ( a, b, c, d, w[ 0], 7, 3614090360); /* 1 */ FF ( d, a, b, c, w[ 1], 12, 3905402710); /* 2 */
        FF ( c, d, a, b, w[ 2], 17,  606105819); /* 3 */FF ( b, c, d, a, w[ 3], 22, 3250441966); /* 4 */
        FF ( a, b, c, d, w[ 4], 7, 4118548399); /* 5 */ FF ( d, a, b, c, w[ 5], 12, 1200080426); /* 6 */
        FF ( c, d, a, b, w[ 6], 17, 2821735955); /* 7 */FF ( b, c, d, a, w[ 7], 22, 4249261313); /* 8 */
        FF ( a, b, c, d, w[ 8], 7, 1770035416); /* 9 */ FF ( d, a, b, c, w[ 9], 12, 2336552879); /* 10 */
        FF ( c, d, a, b, w[10], 17, 4294925233); /* 11 */FF ( b, c, d, a, w[11], 22, 2304563134); /* 12 */
        FF ( a, b, c, d, w[12], 7, 1804603682); /* 13 */FF ( d, a, b, c, w[13], 12, 4254626195); /* 14 */
        FF ( c, d, a, b, w[14], 17, 2792965006); /* 15 */FF ( b, c, d, a, w[15], 22, 1236535329); /* 16 */
        GG ( a, b, c, d, w[ 1], 5, 4129170786); /* 17 */GG ( d, a, b, c, w[ 6], 9, 3225465664); /* 18 */
        GG ( c, d, a, b, w[11], 14,  643717713); /* 19 */GG ( b, c, d, a, w[ 0], 20, 3921069994); /* 20 */
        GG ( a, b, c, d, w[ 5], 5, 3593408605); /* 21 */GG ( d, a, b, c, w[10], 9,   38016083); /* 22 */
        GG ( c, d, a, b, w[15], 14, 3634488961); /* 23 */GG ( b, c, d, a, w[ 4], 20, 3889429448); /* 24 */
        GG ( a, b, c, d, w[ 9], 5,  568446438); /* 25 */ GG ( d, a, b, c, w[14], 9, 3275163606); /* 26 */
        GG ( c, d, a, b, w[ 3], 14, 4107603335); /* 27 */GG ( b, c, d, a, w[ 8], 20, 1163531501); /* 28 */
        GG ( a, b, c, d, w[13], 5, 2850285829); /* 29 */GG ( d, a, b, c, w[ 2], 9, 4243563512); /* 30 */
        GG ( c, d, a, b, w[ 7], 14, 1735328473); /* 31 */GG ( b, c, d, a, w[12], 20, 2368359562); /* 32 */
        HH ( a, b, c, d, w[ 5], 4, 4294588738); /* 33 */HH ( d, a, b, c, w[ 8], 11, 2272392833); /* 34 */
        HH ( c, d, a, b, w[11], 16, 1839030562); /* 35 */HH ( b, c, d, a, w[14], 23, 4259657740); /* 36 */
        HH ( a, b, c, d, w[ 1], 4, 2763975236); /* 37 */HH ( d, a, b, c, w[ 4], 11, 1272893353); /* 38 */
        HH ( c, d, a, b, w[ 7], 16, 4139469664); /* 39 */HH ( b, c, d, a, w[10], 23, 3200236656); /* 40 */
        HH ( a, b, c, d, w[13], 4,  681279174); /* 41 */HH ( d, a, b, c, w[ 0], 11, 3936430074); /* 42 */
        HH ( c, d, a, b, w[ 3], 16, 3572445317); /* 43 */HH ( b, c, d, a, w[ 6], 23,   76029189);/* 44 */
        HH ( a, b, c, d, w[ 9], 4, 3654602809); /* 45 */HH ( d, a, b, c, w[12], 11, 3873151461); /* 46 */
        HH ( c, d, a, b, w[15], 16,  530742520); /* 47 */HH ( b, c, d, a, w[ 2], 23, 3299628645); /* 48 */
        II ( a, b, c, d, w[ 0], 6, 4096336452); /* 49 */II ( d, a, b, c, w[ 7], 10, 1126891415); /* 50 */
        II ( c, d, a, b, w[14], 15, 2878612391); /* 51 */II ( b, c, d, a, w[ 5], 21, 4237533241); /* 52 */
        II ( a, b, c, d, w[12], 6, 1700485571); /* 53 */II ( d, a, b, c, w[ 3], 10, 2399980690); /* 54 */
        II ( c, d, a, b, w[10], 15, 4293915773); /* 55 */II ( b, c, d, a, w[ 1], 21, 2240044497); /* 56 */
        II ( a, b, c, d, w[ 8], 6, 1873313359); /* 57 */II ( d, a, b, c, w[15], 10, 4264355552); /* 58 */
        II ( c, d, a, b, w[ 6], 15, 2734768916); /* 59 */II ( b, c, d, a, w[13], 21, 1309151649); /* 60 */
        II ( a, b, c, d, w[ 4], 6, 4149444226); /* 61 */II ( d, a, b, c, w[11], 10, 3174756917); /* 62 */
        II ( c, d, a, b, w[ 2], 15,  718787259); /* 63 */II ( b, c, d, a, w[ 9], 21, 3951481745); /* 64 */
        // Add this chunk's hash to result so far:
        A += a;
        B += b;
        C += c;
        D += d;
    }

    // cleanup
    free(msg);
    //var char digest[16] := A append B append C append D //(Output is in little-endian)
    cuda_to_bytes(A, digest);
    cuda_to_bytes(B, digest + 4);
    cuda_to_bytes(C, digest + 8);
    cuda_to_bytes(D, digest + 12);
}

// Helper function for using CUDA to compute MD5 with timing
int MD5(const uint8_t *initial_msg, size_t initial_len, uint8_t *digest, int runs)
{
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
    uint8_t *dev_initial_msg = 0;
    uint8_t *dev_digest = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    gpuErrchk(hipSetDevice(0));
    gpuErrchk(hipMalloc((void**)&dev_digest, md5_size * sizeof(uint8_t)));
    gpuErrchk(hipMalloc((void**)&dev_initial_msg, initial_len * sizeof(uint8_t)));
    // Copy input vectors from host memory to GPU buffers.
    gpuErrchk(hipMemcpy(dev_initial_msg, initial_msg, initial_len * sizeof(uint8_t), hipMemcpyHostToDevice));
	gettimeofday(&t_start, NULL);
    for (int i = 0; i < runs; i++) {
        md5kernel<<<1, 1>>>(dev_initial_msg, initial_len, dev_digest);
    }
    gpuErrchk(hipDeviceSynchronize());
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / runs;
        double GBsec = initial_len * (2*sizeof(int) + sizeof(int)) * 1.0e-3f / elapsed;
        printf("GPU2 runs in:       %lu microsecs,   GB/sec: %.2f\n", elapsed, GBsec);

    // Check for any errors launching the kernel
    gpuErrchk(hipGetLastError());
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.

    // Copy output vector from GPU buffer to host memory.
    gpuErrchk(hipMemcpy(digest, dev_digest, md5_size * sizeof(uint8_t), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(dev_digest));
    gpuErrchk(hipFree(dev_initial_msg));

    return 0;
}



int Many_MD5(const uint8_t *initial_msg, size_t initial_len, uint8_t *digest, int runs)
{
    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
    uint8_t *dev_initial_msg = 0;
    uint8_t *dev_digest = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    gpuErrchk(hipSetDevice(0));

    gpuErrchk(hipMalloc((void**)&dev_digest, md5_size * sizeof(uint8_t)));
    gpuErrchk(hipMalloc((void**)&dev_initial_msg, initial_len * sizeof(uint8_t)));
    // Copy input vectors from host memory to GPU buffers.
    gpuErrchk(hipMemcpy(dev_initial_msg, initial_msg, initial_len * sizeof(uint8_t), hipMemcpyHostToDevice));

    // Launch a kernel on the GPU with one thread for each element.
	gettimeofday(&t_start, NULL);
    for (int i = 0; i < runs; i++) {
        many_md5kernel<<<512, 1>>>(dev_initial_msg, initial_len, dev_digest);
    }
    gpuErrchk(hipDeviceSynchronize());
        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec) / runs;
        double GBsec = initial_len * (2*sizeof(int) + sizeof(int)) * 1.0e-3f / elapsed;
        printf("GPU2 runs in:       %lu microsecs,   GB/sec: %.2f\n", elapsed, GBsec);

    // Check for any errors launching the kernel
    gpuErrchk(hipGetLastError());
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.

    // Copy output vector from GPU buffer to host memory.
    gpuErrchk(hipMemcpy(digest, dev_digest, md5_size * sizeof(uint8_t), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(dev_digest));
    gpuErrchk(hipFree(dev_initial_msg));

    return 0;
}
